#include "tplcpp/dyn_prog/common.cuh"

void checkCudaError(hipError_t err, std::string msg) {

    if (hipSuccess != err) {
        throw std::runtime_error(
                std::string(hipGetErrorString(err))
                + ": " + msg);
    }
}
